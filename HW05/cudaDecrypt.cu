#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

__device__ unsigned int CUDAmodProd(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int za = a;
  unsigned int ab = 0;
  while(b > 0) {
    if (b%2 == 1) ab = (ab+za)%p;
    za = (2*za)%p;
    b /= 2;
  }
  return ab;
}

__device__ unsigned int CUDAmodExp(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int z = a;
  unsigned int aExpb = 1;
  while(b > 0) {
    if (b%2==1) aExpb = CUDAmodProd(aExpb, z, p);
    z = CUDAmodProd(z, z, p);
    b /= 2;
  }
  return aExpb;
}

__global__ kernelFindKey(volatile unsigned int *secretKey, unsigned int p, unsigned int g, unsigned int h){
  
  int threadID = threadIdx.x;
  int blockID = blockIdx.x
  int Nblock = blockDim.x;

  unsigned int ID = threadID + blockID*Nblock;
  secretKey[ID+1] = 0;

  if ((ID < p) && CUDAmodExp(g, ID+1, p) == h) {
    
    secretKey[ID+1] = ID+1;
    printf("Secret Key Found %d %d %d\n", ID+1, secretKey[ID+1], secretKey[ID]);
  }

}

int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */

  FILE *file1;
  if ((file1 = fopen("public_key.txt", "r")) == NULL) {
    printf("Error!\n");
    exit(1);
  }

  fscanf(file1, "%u\n", &n);
  fscanf(file1, "%u\n", &p);
  fscanf(file1, "%u\n", &g);
  fscanf(file1, "%u\n", &h);
  fclose(file1);

  FILE *file2;
  if ((file2 = fopen("message.txt", "r")) == NULL) {
    printf("Error!\n");
    exit(1);
  }

  fscanf(file2, "%u\n", &Nints);

  unsigned int count = 0;
  unsigned int *m = (unsigned int *) malloc(Nints*sizeof(unsigned int));
  unsigned int *a = (unsigned int *) malloc(Nints*sizeof(unsigned int));
  while (count < Nints) {
    fscanf(file2, "%u %u\n", &m[count], &a[count]);
    count = count + 1;
  }
  fclose(file2);

  // find the secret key
  if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    double startTime = clock();
    for (unsigned int i=0;i<p-1;i++) {
      if (modExp(g,i+1,p)==h) {
        printf("Secret key found! x = %u \n", i+1);
        x=i+1;
      }
    }
    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  }
  

  double start = clock();
  unsigned int *secretKey;
  hipMalloc(&secretKey, (p-1)*sizeof(unsigned int));
  int Nthreads = 128;
  int Nblocks = (p+Nthreads-1)/Nthreads;
  kernelFindKey <<<Nblocks, Nthreads>>> (secretKey, p, g, h);
  hipDeviceSynchronize();
  unsigned int *final = (unsigned int*) malloc((p-1)*sizeof(unsigned int));
  hipMemcpy(final, secretKey, (p-1)*sizeof(unsigned int), hipMemcpyDeviceToHost);
  
  int w = 0;
  while (w < p) {
    if (final[w] != 0) {
      x = final[w];
      break;
    }
    w++;
  }

  printf("The secret key is %d\n", x);
  double end = clock();
  double total = (end-start)/CLOCKS_PER_SEC;
  double cudawork = (double) p;
  double cudathroughput = cudawork/total;

  printf("Searching all keys with cuda took %g seconds, throughput was %g values tested per second.\n", total, cudathroughput);

  /* Q3 After finding the secret key, decrypt the message */

  ElGamalDecrypt(m, a, Nints, p, x);

  int bufferSize = 1024;
  unsigned char *message = (unsigned char *) malloc(bufferSize*sizeof(unsigned char));
  unsigned int charsPerInt = (int) ((n-1)/8);
  unsigned int Nchars = (Nints * charsPerInt);

  convertZToString(m, Nints, message, Nchars);
  printf("Decrypted Message = \"%s\"\n", message);
  

  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */
  
  return 0;
} 
